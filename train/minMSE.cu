#include "hip/hip_runtime.h"
#include "minMSE.h"


__global__ void mseKernel(unsigned char * posXGrayImage, unsigned char * negXGrayImage, unsigned char *feaPoints,
	unsigned char *NPDtable, float *posXweight, float *negXweight, int posX_size, int negX_size, int minLeaf,
	float *mse, float *fit0, float * fit1, int *thr0, int *thr1)
{
	const unsigned int  feaId = (blockIdx.x*blockDim.x) + threadIdx.x;
	
	unsigned char  x_1 = feaPoints[feaId * 4 + 0];
	unsigned char y_1 = feaPoints[feaId * 4 + 1];
	unsigned char x_2 = feaPoints[feaId * 4 + 2];
	unsigned char y_2 = feaPoints[feaId * 4 + 3];
	int count[256];
	float posWHist[256];
	float negWHist[256];
	for (int i = 0; i < 256; i++)
	{
		count[i] = 0;
		posWHist[i] = 0;
		negWHist[i] = 0;
	}
	for (int i = 0; i < posX_size; i++)
	{
		unsigned char *image = posXGrayImage + i*SAMPLE_SIZE*SAMPLE_SIZE;
		unsigned char fea = NPDtable[image[y_1*SAMPLE_SIZE + x_1] * GRAY_LEVEL +
			image[y_2*SAMPLE_SIZE + x_2]];
		posWHist[fea] += posXweight[i];
		count[fea]++;

	}
	for (int i = 0; i < negX_size; i++)
	{
		unsigned char * image = negXGrayImage + i*SAMPLE_SIZE*SAMPLE_SIZE;
		unsigned char fea = NPDtable[image[y_1*SAMPLE_SIZE + x_1] * GRAY_LEVEL +
			image[y_2*SAMPLE_SIZE + x_2]];
		negWHist[fea] += negXweight[i];
		count[fea]++;

	}

	float posWSum = 0.0;
	float negWSum = 0.0;
	for (int bin = 0; bin < 256; bin++)
	{
		posWSum += posWHist[bin];
		negWSum += negWHist[bin];
	}

	int totalCount = posX_size + negX_size;
	float wSum = posWSum + negWSum;
	float minMSE = FLT_MAX;
	int localThr0 = -1, localThr1;
	float localFit0, localFit1;

	for (int v = 0; v < 256; v++) // lower threshold
	{
		int rightCount = 0;
		float rightPosW = 0;
		float rightNegW = 0;
		for (int u = v; u < 256; u++) // upper threshold
		{
			rightCount += count[u];
			rightPosW += posWHist[u];
			rightNegW += negWHist[u];
			if (rightCount < minLeaf)
			{
				continue;
			}
			int leftCount = totalCount - rightCount;
			if (leftCount < minLeaf)
			{
				break;
			}
			float leftPosW = posWSum - rightPosW;
			float leftNegW = negWSum - rightNegW;
			float leftFit, rightFit;
			if (leftPosW + leftNegW <= 0)
			{
				leftFit = 0.0f;
			}
			else
			{
				leftFit = (leftPosW - leftNegW) / (leftPosW + leftNegW);
			}

			if (rightPosW + rightNegW <= 0)
			{
				rightFit = 0.0f;
			}
			else
			{
				rightFit = (rightPosW - rightNegW) / (rightPosW + rightNegW);
			}
			float leftMSE = leftPosW * (leftFit - 1) * (leftFit - 1) +
				leftNegW * (leftFit + 1) * (leftFit + 1);
			float rightMSE = rightPosW * (rightFit - 1) * (rightFit - 1) +
				rightNegW * (rightFit + 1) * (rightFit + 1);
			float localMSE = leftMSE + rightMSE;
			if (localMSE < minMSE)
			{
				minMSE = localMSE;
				localThr0 = v;
				localThr1 = u;
				localFit0 = leftFit;
				localFit1 = rightFit;
			}
		}
	}
	if (localThr0 == -1)
	{
		mse[feaId] = FLT_MAX;
	}
	else
	{
		mse[feaId] = minMSE;
		fit0[feaId] = localFit0;
		fit1[feaId] = localFit1;
		thr0[feaId] = localThr0;
		thr1[feaId] = localThr1;
	}

}

void gpuMSE(DQTreeNode ** root_, std::vector<OneSample*>& posX,
	std::vector<OneSample*>& negX, const Configuration & config,
	float *fit, float minCost)
{
	/**********************************************************************************************/
	uchar * posXArray_local = new unsigned char[SAMPLE_SIZE*SAMPLE_SIZE*posX.size()];
	float * posXWeight_local = new float[posX.size()];
	for (int i = 0; i < posX.size(); i++)
	{
		memcpy(posXArray_local + SAMPLE_SIZE*SAMPLE_SIZE*i, posX[i]->grayImage,
			sizeof(uchar)*SAMPLE_SIZE*SAMPLE_SIZE);
		posXWeight_local[i] = posX[i]->weight;
	}
	
	//
	uchar * posXArray_dev = NULL;
	hipMalloc((void **)&posXArray_dev, sizeof(uchar)*SAMPLE_SIZE*SAMPLE_SIZE*posX.size());
	hipMemcpy(posXArray_dev, posXArray_local,
		sizeof(uchar)*SAMPLE_SIZE*SAMPLE_SIZE*posX.size(), hipMemcpyHostToDevice);
	//
	float * posXWeight_dev = NULL;
	hipMalloc((void **)&posXWeight_dev, sizeof(float)*posX.size());
	hipMemcpy(posXWeight_dev, posXWeight_local,
		sizeof(float)*posX.size(), hipMemcpyHostToDevice);
	/**********************************************************************************************/
	uchar * negXArray_local = new unsigned char[SAMPLE_SIZE*SAMPLE_SIZE*negX.size()];
	float * negXWeight_local = new float[negX.size()];
	for (int i = 0; i < negX.size(); i++)
	{
		memcpy(negXArray_local + SAMPLE_SIZE*SAMPLE_SIZE*i, negX[i]->grayImage,
			sizeof(uchar)*SAMPLE_SIZE*SAMPLE_SIZE);
		negXWeight_local[i] = negX[i]->weight;
	}
	//for (int pixel_i = 0; pixel_i < SAMPLE_SIZE*SAMPLE_SIZE; pixel_i++)
	//{
	//	printf("pixel1:%d pixel2:%d\n", (int)(negXArray_local + SAMPLE_SIZE*SAMPLE_SIZE*(negX.size() - 1))[pixel_i],
	//		(int)(negX[(negX.size() - 1)]->grayImage)[0][pixel_i]);
	//}
	//
	uchar * negXArray_dev = NULL;
	hipMalloc((void **)&negXArray_dev, sizeof(uchar)*SAMPLE_SIZE*SAMPLE_SIZE*negX.size());
	hipMemcpy(negXArray_dev, negXArray_local,
		sizeof(uchar)*SAMPLE_SIZE*SAMPLE_SIZE*negX.size(), hipMemcpyHostToDevice);
	//
	float * negXWeight_dev = NULL;
	hipMalloc((void **)&negXWeight_dev, sizeof(float)*negX.size());
	hipMemcpy(negXWeight_dev, negXWeight_local,
		sizeof(float)*negX.size(), hipMemcpyHostToDevice);
	/**********************************************************************************************/
	//
	uchar *feaPoints_dev = NULL;
	hipMalloc((void **)&feaPoints_dev, sizeof(uchar)*FEATURE_SIZE*4);
	hipMemcpy(feaPoints_dev, npdFea.feaPoints,
		sizeof(uchar)*FEATURE_SIZE*4, hipMemcpyHostToDevice);
	//
	uchar *NPDtable_dev = NULL;
	hipMalloc((void **)&NPDtable_dev, sizeof(uchar)*GRAY_LEVEL*GRAY_LEVEL);
	hipMemcpy(NPDtable_dev, npdFea.NPDtable,
		sizeof(uchar)*GRAY_LEVEL*GRAY_LEVEL, hipMemcpyHostToDevice);
	//unsigned char * posXGrayImage,unsigned char * negXGrayImage, unsigned char *feaPoints,
	//	unsigned char *NPDtable, float *posXweight,float *negXweight, int posX_size, int negX_size, int minLeaf,
	//	float *mse, float *fit0, float * fit1, float *thr0, float *thr1

	float *mse_dev = NULL;
	hipMalloc((void **)&mse_dev, sizeof(float)*FEATURE_SIZE);
	float *fit0_dev = NULL;
	hipMalloc((void **)&fit0_dev, sizeof(float)*FEATURE_SIZE);
	float * fit1_dev = NULL;
	hipMalloc((void **)&fit1_dev, sizeof(float)*FEATURE_SIZE);
	int *thr0_dev = NULL;
	hipMalloc((void **)&thr0_dev, sizeof(int)*FEATURE_SIZE);
	int *thr1_dev = NULL;
	hipMalloc((void **)&thr1_dev, sizeof(int)*FEATURE_SIZE);


	//dim3 thread_rect(SAMPLE_SIZE/2, SAMPLE_SIZE);
	//dim3 block_rect();
	mseKernel<<<SAMPLE_SIZE*SAMPLE_SIZE - 1, SAMPLE_SIZE*SAMPLE_SIZE / 2 >>>
		(posXArray_dev, negXArray_dev, feaPoints_dev, NPDtable_dev, posXWeight_dev,
		negXWeight_dev,posX.size(), negX.size(),  config.minLeaf,mse_dev, fit0_dev,
		fit1_dev, thr0_dev, thr1_dev);
	float* mse = new float[FEATURE_SIZE];
	float* fit0 = new float[FEATURE_SIZE];
	float* fit1 = new float[FEATURE_SIZE];
	int* thr0 = new int[FEATURE_SIZE];
	int* thr1 = new int[FEATURE_SIZE];

	hipMemcpy(mse, mse_dev, sizeof(float)*FEATURE_SIZE, hipMemcpyDeviceToHost);
	hipMemcpy(fit0, fit0_dev, sizeof(float)*FEATURE_SIZE, hipMemcpyDeviceToHost);
	hipMemcpy(fit1, fit1_dev, sizeof(float)*FEATURE_SIZE, hipMemcpyDeviceToHost);
	hipMemcpy(thr0, thr0_dev, sizeof(int)*FEATURE_SIZE, hipMemcpyDeviceToHost);
	hipMemcpy(thr1, thr1_dev, sizeof(int)*FEATURE_SIZE, hipMemcpyDeviceToHost);

	
	
	findMinMSE(root_, fit, mse, thr0, thr1, fit0, fit1, minCost);

	delete mse;
	delete fit0;
	delete fit1;
	delete thr0;
	delete thr1;

	delete posXArray_local;
	delete negXArray_local;
	delete posXWeight_local;
	delete negXWeight_local;
	

	hipFree(posXArray_dev);
	hipFree(negXArray_dev);
	hipFree(posXWeight_dev);
	hipFree(negXWeight_dev);
	hipFree(feaPoints_dev);
	hipFree(NPDtable_dev);
	

	hipFree(mse_dev);
	hipFree(fit0_dev);
	hipFree(fit1_dev);
	hipFree(thr0_dev);
	hipFree(thr1_dev);


}
